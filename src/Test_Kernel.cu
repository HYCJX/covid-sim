#include "hip/hip_runtime.h"
#include "Test_Kernel.cuh"
#include "Sweep_gpu_Helper.cuh"

// Error Handling.
void handle_error(hipError_t error) {
    if (error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
}

/* ----- Test Cell ----- */

__global__ void kernel_test_Cell(Cell *c, int j) {
    printf("GPU: c->I = %d.\n", c->I);
    printf("GPU: c->infected[%d] = %d.\n", j, c->infected[j]);
}

void test_Cell(Cell *c, int j) {
    struct Cell *c_GPU;
    struct Cell *c_Builder = (Cell *) malloc(sizeof(Cell));
    c_Builder->I = c->I;
    handle_error(hipMalloc((void **) &c_Builder->infected, c->I * sizeof(int)));
    handle_error(hipMemcpy(c_Builder->infected, c->infected, c->I * sizeof(int), hipMemcpyHostToDevice));
    handle_error(hipMalloc((void **) &c_GPU, sizeof(struct Cell)));
    handle_error(hipMemcpy(c_GPU, c_Builder, sizeof(struct Cell), hipMemcpyHostToDevice));

    kernel_test_Cell<<<1,1>>>(c_GPU, j);
    handle_error(hipDeviceSynchronize());

    handle_error(hipFree(c_Builder->infected));
    handle_error(hipFree(c_GPU));
    free(c_Builder);
}

/* ----- Test Hosts ----- */

__global__ void kernel_test_Hosts(Person *Hosts_GPU, int index, int place_index, int place_group_index) {
    if (place_index >= 0) {
        printf("GPU: Hosts[%d].PlaceLinks[%d] = %d.\n", index, place_index, (Hosts_GPU + index) -> PlaceLinks[place_index]);
    }
    if (place_group_index >= 0){
        printf("GPU: Hosts[%d].PlaceGroupLinks[%d] = %d.\n", index, place_group_index, (Hosts_GPU + index) -> PlaceGroupLinks[place_group_index]);
    }

}

void test_Hosts(int index, int place_index, int place_group_index) {
    struct Person *Hosts_GPU;
    handle_error(hipMalloc((void **) &Hosts_GPU, P.PopSize * sizeof(struct Person)));
    handle_error(hipMemcpy(Hosts_GPU, Hosts, P.PopSize * sizeof(struct Person), hipMemcpyHostToDevice));
    kernel_test_Hosts<<<1,1>>>(Hosts_GPU, index, place_index, place_group_index);
    handle_error(hipDeviceSynchronize());
    handle_error(hipMemcpy(Hosts, Hosts_GPU, P.PopSize * sizeof(struct Person), hipMemcpyDeviceToHost));
    handle_error(hipFree(Hosts_GPU));
}

/* ----- Test HostsQuarantine ----- */

__global__ void kernel_test_HostsQuarantine(PersonQuarantine *HostsQuarantine_GPU, int index) {
    printf("GPU: HostsQuarantine[%d].comply = %d.\n", index, HostsQuarantine_GPU[index].comply);
    printf("GPU: HostsQuarantine[%d].start_time = %d.\n", index, HostsQuarantine_GPU[index].start_time);
}

void test_HostsQuarantine(int index) {
    struct PersonQuarantine *HostsQuarantine_GPU;
    handle_error(hipMalloc((void **) &HostsQuarantine_GPU, HostsQuarantine.size() * sizeof(struct PersonQuarantine)));
    handle_error(hipMemcpy(HostsQuarantine_GPU, &HostsQuarantine[0], HostsQuarantine.size() * sizeof(struct PersonQuarantine), hipMemcpyHostToDevice));
    kernel_test_HostsQuarantine<<<1,1>>>(HostsQuarantine_GPU, index);
    handle_error(hipDeviceSynchronize());
    handle_error(hipFree(HostsQuarantine_GPU));
}

/* ----- Test Households ----- */

__global__ void kernel_test_Households(Household *Households_GPU, int index) {
    printf("GPU: Households[%d].FirstPerson = %d.\n", index, Households_GPU[index].FirstPerson);
    printf("GPU: Households[%d].nh = %u.\n", index, Households_GPU[index].nh);
    printf("GPU: Households[%d].nhr = %u.\n", index, Households_GPU[index].nhr);
    printf("GPU: Households[%d].loc.x = %9.6f.\n", index, Households_GPU[index].loc.x);
    printf("GPU: Households[%d].loc.y = %9.6f.\n", index, Households_GPU[index].loc.y);
}

void test_HouseHolds(int index) {
    struct Household *Households_GPU;
    handle_error(hipMalloc((void **) &Households_GPU, P.NH * sizeof(struct Household)));
    handle_error(hipMemcpy(Households_GPU, Households, P.NH * sizeof(struct Household), hipMemcpyHostToDevice));
    kernel_test_Households<<<1,1>>>(Households_GPU, index);
    handle_error(hipDeviceSynchronize());
    handle_error(hipFree(Households_GPU));
}

/* ----- Test Mcells ----- */

__global__ void kernel_test_Mcells(Microcell *Mcells_GPU, int index) {
    printf("GPU: Mcells[%d].adunit = %d.\n", index, Mcells_GPU[index].adunit);
    printf("GPU: Mcells[%d].moverest = %u.\n", index, Mcells_GPU[index].moverest);
    printf("GPU: Mcells[%d].socdist = %u.\n", index, Mcells_GPU[index].socdist);
}

void test_Mcells(int index) {
    struct Microcell *Mcells_GPU;
    handle_error(hipMalloc((void **) &Mcells_GPU, P.NMC * sizeof(struct Microcell)));
    handle_error(hipMemcpy(Mcells_GPU, Mcells, P.NMC * sizeof(struct Microcell), hipMemcpyHostToDevice));
    kernel_test_Mcells<<<1,1>>>(Mcells_GPU, index);
    handle_error(hipDeviceSynchronize());
    handle_error(hipFree(Mcells_GPU));
}

/* ----- Test Places -----*/

__global__ void kernel_test_places(Place **Places_GPU, int i, int j, int group_start_i, int group_size_i, int members_i) {
    printf("GPU: Places[%d][%d].n = %d.\n", i, j, Places_GPU[i][j].n);
    printf("GPU: Places[%d][%d].mcell = %d.\n", i, j, Places_GPU[i][j].mcell);
    printf("GPU: Places[%d][%d].loc.x = %9.6f.\n", i, j, Places_GPU[i][j].loc.x);
    printf("GPU: Places[%d][%d].loc.y = %9.6f.\n", i, j, Places_GPU[i][j].loc.y);
    if (group_start_i >= 0) {
        printf("GPU: Places[%d][%d].group_start[%d] = %d.\n", i, j, group_start_i, Places_GPU[i][j].group_start[group_start_i]);
    }
    if (group_size_i >= 0) {
        printf("GPU: Places[%d][%d].group_size[%d] = %d.\n", i, j, group_size_i, Places_GPU[i][j].group_size[group_size_i]);
    }
    if (members_i >= 0) {
        printf("GPU: Places[%d][%d].members[%d] = %d.\n", i, j, members_i, Places_GPU[i][j].members[members_i]);
    }
}

void test_Places(int i, int j, int group_start_i, int group_size_i, int members_i){
    /* --- Start Time Record --- */
    hipEvent_t start, stop;
    handle_error(hipEventCreate(&start));
    handle_error(hipEventCreate(&stop));
    handle_error(hipEventRecord(start, 0));
    /* ---                   --- */

    struct Place **Struct_Builder = (struct Place **) malloc(P.PlaceTypeNum * sizeof(struct Place *));
    for (int p = 0; p < P.PlaceTypeNum; p++) {
        Struct_Builder[p] = (struct Place *) malloc(P.Nplace[p] * sizeof(struct Place));
        for (int q = 0; q < P.Nplace[p]; q++) {
            Place place = Places[p][q];
            Struct_Builder[p][q] = place;
            handle_error(hipMalloc((void **) &Struct_Builder[p][q].group_start, place.ng * sizeof(int)));
            handle_error(hipMemcpy(Struct_Builder[p][q].group_start, place.group_start, place.ng * sizeof(int), hipMemcpyHostToDevice));
            handle_error(hipMalloc((void **) &Struct_Builder[p][q].group_size, place.ng * sizeof(int)));
            handle_error(hipMemcpy(Struct_Builder[p][q].group_size, place.group_size, place.ng * sizeof(int), hipMemcpyHostToDevice));
            if (p == P.HotelPlaceType) {
                handle_error(hipMalloc((void **) &Struct_Builder[p][q].members, 2 * ((int)P.PlaceTypeMeanSize[p]) * sizeof(int)));
                handle_error(hipMemcpy(Struct_Builder[p][q].members, place.members, 2 * ((int)P.PlaceTypeMeanSize[p]) * sizeof(int), hipMemcpyHostToDevice));
            } else {
                handle_error(hipMalloc((void **) &Struct_Builder[p][q].members, place.n * sizeof(int)));
                handle_error(hipMemcpy(Struct_Builder[p][q].members, place.members, place.n * sizeof(int), hipMemcpyHostToDevice));
            }
        }
    }
    struct Place **Places_GPU;
    struct Place *Places_Builder[P.PlaceTypeNum];
    handle_error(hipMalloc((void **) &Places_GPU, P.PlaceTypeNum * sizeof(struct Place *)));
    for (int m = 0; m < P.PlaceTypeNum; m++) {
        handle_error(hipMalloc((void **) &Places_Builder[m], P.Nplace[m] * sizeof(struct Place)));
        handle_error(hipMemcpy(Places_Builder[m], Struct_Builder[m], P.Nplace[m] * sizeof(struct Place), hipMemcpyHostToDevice));
    }
    handle_error(hipMemcpy(Places_GPU, Places_Builder, P.PlaceTypeNum * sizeof(struct Place *),hipMemcpyHostToDevice));

//    kernel_test_places<<<1,1>>>(Places_GPU, i, j, group_start_i, group_size_i, members_i);
//    handle_error(hipDeviceSynchronize());

    for (int m = 0; m < P.PlaceTypeNum; m++) {
        for (int n = 0; n < P.Nplace[m]; n++) {
            handle_error(hipFree(Struct_Builder[m][n].group_start));
            handle_error(hipFree(Struct_Builder[m][n].group_size));
            handle_error(hipFree(Struct_Builder[m][n].members));
        }
        free(Struct_Builder[m]);
    }
    free(Struct_Builder);
    for (int i = 0; i < P.PlaceTypeNum; i++) {
        handle_error(hipFree(Places_Builder[i]));
    }
    handle_error(hipFree(Places_GPU));
    /* --- Stop Time Record --- */
    handle_error(hipEventRecord(stop, 0));
    handle_error(hipEventSynchronize(stop));
    float elapsedTime;
    handle_error(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Data Transfer Time: %3.lf ms.\n", elapsedTime);
    handle_error(hipEventDestroy(start));
    handle_error(hipEventDestroy(stop));
    /* ---                  --- */
}

/* ----- Test AdUnits ----- */

__global__ void kernel_test_AdUnits(AdminUnit *AdUnits_GPU, int index) {
    printf("GPU: AdUnits[%d].n = %d.\n", index, AdUnits_GPU[index].n);
    printf("GPU: AdUnits[%d].DigitalContactTracingTimeStart = %lf.\n", index, AdUnits_GPU[index].DigitalContactTracingTimeStart);
}

void test_AdUnits(int index) {
    struct AdminUnit *AdUnits_GPU;
    handle_error(hipMalloc((void **) &AdUnits_GPU, MAX_ADUNITS * sizeof(struct AdminUnit)));
    handle_error(hipMemcpy(AdUnits_GPU, AdUnits, MAX_ADUNITS * sizeof(struct AdminUnit), hipMemcpyHostToDevice));

    kernel_test_AdUnits<<<1,1>>>(AdUnits_GPU, index);

    handle_error(hipFree(AdUnits_GPU));
}

/* ----- Test SamplingQueue ----- */

__global__ void kernel_test_SamplingQueue(int **SamplingQueue_GPU, int i, int j) {
    printf("GPU: SamplingQueue[%d][%d] = %d.\n", i, j, SamplingQueue_GPU[i][j]);
}

void test_SamplingQueue(int i, int j) {
    int **SamplingQueue_GPU;
    int *SamplingQueue_Builder[P.NumThreads];
    handle_error(hipMalloc((void **) &SamplingQueue_GPU, P.NumThreads * sizeof(int *)));
    for (int i = 0; i < P.NumThreads; i++) {
        handle_error(hipMalloc((void **) &SamplingQueue_Builder[i],2 * (MAX_PLACE_SIZE + CACHE_LINE_SIZE) * sizeof(int)));
        handle_error(hipMemcpy(SamplingQueue_Builder[i], SamplingQueue[i],2 * (MAX_PLACE_SIZE + CACHE_LINE_SIZE) * sizeof(int), hipMemcpyHostToDevice));
    }
    handle_error(hipMemcpy(SamplingQueue_GPU, SamplingQueue_Builder, P.NumThreads * sizeof(int *),hipMemcpyHostToDevice));

    kernel_test_SamplingQueue<<<1,1>>>(SamplingQueue_GPU, i, j);
    handle_error(hipDeviceSynchronize());

    handle_error(hipMemcpy(SamplingQueue_Builder, SamplingQueue_GPU, P.NumThreads * sizeof(int *),hipMemcpyDeviceToHost));
    for (int i = 0; i < P.NumThreads; i++) {
        handle_error(hipMemcpy(SamplingQueue[i], SamplingQueue_Builder[i], 2 * (MAX_PLACE_SIZE + CACHE_LINE_SIZE) * sizeof(int), hipMemcpyDeviceToHost));
        handle_error(hipFree(SamplingQueue_Builder[i]));
    }
    hipFree(SamplingQueue_GPU);
}

/* ----- Test StateT ----- */

__global__ void kernel_test_StateT(PopVar *StateT_GPU, int index, int n_queue_index, int cell_inf_index) {
    if (n_queue_index >= 0) {
        printf("GPU: StateT[%d].n_queue[%d] = %d.\n", index, n_queue_index, StateT_GPU[index].n_queue[n_queue_index]);
    }
    if (cell_inf_index >= 0) {
        printf("GPU: StateT[%d].cell_inf[%d] = %9.6f.\n", index, cell_inf_index, StateT_GPU[index].cell_inf[cell_inf_index]);
    }
    StateT_GPU[index].n_queue[n_queue_index] *= 10;
    StateT_GPU[index].cell_inf[cell_inf_index] *= 10;
}

void test_StateT(int index, int n_queue_index, int cell_inf_index) {
    struct PopVar *StateT_GPU;
    struct PopVar *StateT_Builder = (struct PopVar *) malloc(P.NumThreads * sizeof(struct PopVar));
    memcpy(StateT_Builder, StateT, P.NumThreads * sizeof(struct PopVar));
    for (int i = 0; i < P.NumThreads; i++) {
        for (int j = 0; j < P.NumThreads; j++) {
            handle_error(hipMalloc((void **) &(StateT_Builder[i].inf_queue[j]),StateT[i].n_queue[j] * sizeof(Infection)));
            handle_error(hipMemcpy(StateT_Builder[i].inf_queue[j], StateT[i].inf_queue[j],StateT[i].n_queue[j] * sizeof(Infection),hipMemcpyHostToDevice));
        }
        handle_error(hipMalloc((void **) &StateT_Builder[i].cell_inf, StateT[i].cell_inf_length * sizeof(float)));
        handle_error(hipMemcpy(StateT_Builder[i].cell_inf, StateT[i].cell_inf, StateT[i].cell_inf_length * sizeof(float),hipMemcpyHostToDevice));
        for (int j = 0; j < P.NumAdunits; j++) {
            handle_error(hipMalloc((void **) &(StateT_Builder[i].dct_queue[j]), StateT[i].ndct_queue[j] * sizeof(ContactEvent)));
            handle_error(hipMemcpy(StateT_Builder[i].dct_queue[j], StateT[i].dct_queue[j], StateT[i].ndct_queue[j] * sizeof(ContactEvent), hipMemcpyHostToDevice));
        }
    }
    handle_error(hipMalloc((void **) &StateT_GPU, P.NumThreads * sizeof(struct PopVar)));
    handle_error(hipMemcpy(StateT_GPU, StateT_Builder, P.NumThreads * sizeof(struct PopVar),hipMemcpyHostToDevice));

    kernel_test_StateT<<<1,1>>>(StateT_GPU, index, n_queue_index, cell_inf_index);
    handle_error(hipDeviceSynchronize());

    handle_error(hipMemcpy(StateT_Builder, StateT_GPU, P.NumThreads * sizeof(struct PopVar),hipMemcpyDeviceToHost));
    handle_error(hipFree(StateT_GPU));
    for (int i = 0; i < P.NumThreads; i++) {
        for (int j = 0; j < MAX_NUM_THREADS; j++) {
            handle_error(hipMemcpy(StateT[i].inf_queue[j], StateT_Builder[i].inf_queue[j], StateT[i].n_queue[j] * sizeof(Infection), hipMemcpyDeviceToHost));
            handle_error(hipFree(StateT_Builder[i].inf_queue[j]));
        }
        memcpy(StateT[i].n_queue, StateT_Builder[i].n_queue, MAX_NUM_THREADS * sizeof(int));
        handle_error(hipFree(StateT_Builder[i].cell_inf));
        for (int j = 0; j < P.NumAdunits; j++) {
            handle_error(hipMemcpy(StateT[i].dct_queue[j], StateT_Builder[i].dct_queue[j], StateT[i].ndct_queue[j] * sizeof(ContactEvent), hipMemcpyDeviceToHost));
            handle_error(hipFree(StateT_Builder[i].dct_queue[j]));
        }
    }
    free(StateT_Builder);
}

/*Test Static Data */

__global__ void kernel_test_static_data() {
    printf("GPU: sinx_GPU[%d] = %f.\n", 1, sinx_GPU[1]);
    printf("GPU: cosx_GPU[%d] = %f.\n", 1, cosx_GPU[1]);
    printf("GPU: asin2sqx_GPU[%d] = %f.\n", 1, asin2sqx_GPU[1]);
    printf("GPU: Xcg1_GPU[%d] = %d.\n", 1, Xcg1_GPU[1]);
    printf("GPU: Xcg2_GPU[%d] = %d.\n", 1, Xcg2_GPU[1]);
}

void test_static_data() {
//    handle_error(hipMemcpyToSymbol(HIP_SYMBOL(sinx_GPU), sinx, (DEGREES_PER_TURN + 1) * sizeof(double)));
//    handle_error(hipMemcpyToSymbol(HIP_SYMBOL(cosx_GPU), cosx, (DEGREES_PER_TURN + 1) * sizeof(double)));
//    handle_error(hipMemcpyToSymbol(HIP_SYMBOL(asin2sqx_GPU), asin2sqx, (1001) * sizeof(double)));
//    handle_error(hipMemcpyToSymbol(HIP_SYMBOL(Xcg1_GPU), Xcg1, (MAX_NUM_THREADS * CACHE_LINE_SIZE) * sizeof(int32_t)));
//    handle_error(hipMemcpyToSymbol(HIP_SYMBOL(Xcg2_GPU), Xcg2, (MAX_NUM_THREADS * CACHE_LINE_SIZE) * sizeof(int32_t)));
    kernel_test_static_data<<<1,1>>>();
    handle_error(hipDeviceSynchronize());
}

/* Test P */

__global__ void kernel_test_P() {}

void test_P() {
    struct Param *P_GPU;
    handle_error(hipMalloc((void **) &P_GPU, sizeof(struct Param)));
    handle_error(hipMemcpy(P_GPU, &P, sizeof(struct Param), hipMemcpyHostToDevice));
    handle_error(hipMemcpy(&P, P_GPU, sizeof(struct Param), hipMemcpyDeviceToHost));
    handle_error(hipFree(P_GPU));
}

/* Integration Test */

void test_all(Cell *c) {

    /* --- Start Time Record --- */
    hipEvent_t start, stop;
    handle_error(hipEventCreate(&start));
    handle_error(hipEventCreate(&stop));
    handle_error(hipEventRecord(start, 0));
    /* ---                   --- */

    /* --- Copy Data: Host to Device --- */
    // Dist global variables:
    handle_error(hipMemcpyToSymbol(HIP_SYMBOL(sinx_GPU), sinx, (DEGREES_PER_TURN + 1) * sizeof(double)));
    handle_error(hipMemcpyToSymbol(HIP_SYMBOL(cosx_GPU), cosx, (DEGREES_PER_TURN + 1) * sizeof(double)));
    // Rand global variables:
    handle_error(hipMemcpyToSymbol(HIP_SYMBOL(asin2sqx_GPU), asin2sqx, (1001) * sizeof(double)));
    handle_error(hipMemcpyToSymbol(HIP_SYMBOL(Xcg1_GPU), Xcg1, (MAX_NUM_THREADS * CACHE_LINE_SIZE) * sizeof(int32_t)));
    handle_error(hipMemcpyToSymbol(HIP_SYMBOL(Xcg2_GPU), Xcg2, (MAX_NUM_THREADS * CACHE_LINE_SIZE) * sizeof(int32_t)));
    // Cell:
    struct Cell *c_GPU;
    struct Cell *c_Builder = (Cell *) malloc(sizeof(Cell));
    c_Builder->I = c->I;
    handle_error(hipMalloc((void **) &c_Builder->infected, c->I * sizeof(int)));
    handle_error(hipMemcpy(c_Builder->infected, c->infected, c->I * sizeof(int), hipMemcpyHostToDevice));
    handle_error(hipMalloc((void **) &c_GPU, sizeof(struct Cell)));
    handle_error(hipMemcpy(c_GPU, c_Builder, sizeof(struct Cell), hipMemcpyHostToDevice));
    // Hosts:
    struct Person *Hosts_GPU;
    handle_error(hipMalloc((void **) &Hosts_GPU, P.PopSize * sizeof(struct Person)));
    handle_error(hipMemcpy(Hosts_GPU, Hosts, P.PopSize * sizeof(struct Person), hipMemcpyHostToDevice));
    // HostsQuarantine:
    struct PersonQuarantine *HostsQuarantine_GPU;
    handle_error(hipMalloc((void **) &HostsQuarantine_GPU, HostsQuarantine.size() * sizeof(struct PersonQuarantine)));
    handle_error(hipMemcpy(HostsQuarantine_GPU, &HostsQuarantine[0], HostsQuarantine.size() * sizeof(struct PersonQuarantine), hipMemcpyHostToDevice));
    // Households:
    struct Household *Households_GPU;
    handle_error(hipMalloc((void **) &Households_GPU, P.NH * sizeof(struct Household)));
    handle_error(hipMemcpy(Households_GPU, Households, P.NH * sizeof(struct Household), hipMemcpyHostToDevice));
    // Mcells:
    struct Microcell *Mcells_GPU;
    handle_error(hipMalloc((void **) &Mcells_GPU, P.NMC * sizeof(struct Microcell)));
    handle_error(hipMemcpy(Mcells_GPU, Mcells, P.NMC * sizeof(struct Microcell), hipMemcpyHostToDevice));
    // Places:
    struct Place **Struct_Builder = (struct Place **) malloc(P.PlaceTypeNum * sizeof(struct Place *));
    for (int p = 0; p < P.PlaceTypeNum; p++) {
        Struct_Builder[p] = (struct Place *) malloc(P.Nplace[p] * sizeof(struct Place));
        for (int q = 0; q < P.Nplace[p]; q++) {
            Place place = Places[p][q];
            Struct_Builder[p][q] = place;
            handle_error(hipMalloc((void **) &Struct_Builder[p][q].group_start, place.ng * sizeof(int)));
            handle_error(hipMemcpy(Struct_Builder[p][q].group_start, place.group_start, place.ng * sizeof(int), hipMemcpyHostToDevice));
            handle_error(hipMalloc((void **) &Struct_Builder[p][q].group_size, place.ng * sizeof(int)));
            handle_error(hipMemcpy(Struct_Builder[p][q].group_size, place.group_size, place.ng * sizeof(int), hipMemcpyHostToDevice));
            if (p == P.HotelPlaceType) {
                handle_error(hipMalloc((void **) &Struct_Builder[p][q].members, 2 * ((int)P.PlaceTypeMeanSize[p]) * sizeof(int)));
                handle_error(hipMemcpy(Struct_Builder[p][q].members, place.members, 2 * ((int)P.PlaceTypeMeanSize[p]) * sizeof(int), hipMemcpyHostToDevice));
            } else {
                handle_error(hipMalloc((void **) &Struct_Builder[p][q].members, place.n * sizeof(int)));
                handle_error(hipMemcpy(Struct_Builder[p][q].members, place.members, place.n * sizeof(int), hipMemcpyHostToDevice));
            }
        }
    }
    struct Place **Places_GPU;
    struct Place *Places_Builder[P.PlaceTypeNum];
    handle_error(hipMalloc((void **) &Places_GPU, P.PlaceTypeNum * sizeof(struct Place *)));
    for (int m = 0; m < P.PlaceTypeNum; m++) {
        handle_error(hipMalloc((void **) &Places_Builder[m], P.Nplace[m] * sizeof(struct Place)));
        handle_error(hipMemcpy(Places_Builder[m], Struct_Builder[m], P.Nplace[m] * sizeof(struct Place), hipMemcpyHostToDevice));
    }
    handle_error(hipMemcpy(Places_GPU, Places_Builder, P.PlaceTypeNum * sizeof(struct Place *),hipMemcpyHostToDevice));

    /* ---                           --- */

    /* --- Copy Data: Device to Host & Free Memory --- */
    // Cell:
    handle_error(hipFree(c_Builder->infected));
    handle_error(hipFree(c_GPU));
    free(c_Builder);
    // Hosts:
    handle_error(hipMemcpy(Hosts, Hosts_GPU, P.PopSize * sizeof(struct Person), hipMemcpyDeviceToHost));
    handle_error(hipFree(Hosts_GPU));
    // HostsQuarantine:
    handle_error(hipFree(HostsQuarantine_GPU));
    // Households:
    handle_error(hipFree(Households_GPU));
    // Mcells:
    handle_error(hipFree(Mcells_GPU));
    // Places:
    for (int m = 0; m < P.PlaceTypeNum; m++) {
        for (int n = 0; n < P.Nplace[m]; n++) {
            handle_error(hipFree(Struct_Builder[m][n].group_start));
            handle_error(hipFree(Struct_Builder[m][n].group_size));
            handle_error(hipFree(Struct_Builder[m][n].members));
        }
        free(Struct_Builder[m]);
    }
    free(Struct_Builder);
    for (int i = 0; i < P.PlaceTypeNum; i++) {
        handle_error(hipFree(Places_Builder[i]));
    }
    handle_error(hipFree(Places_GPU));

    /* ---                                         --- */

    /* --- Stop Time Record --- */
    handle_error(hipEventRecord(stop, 0));
    handle_error(hipEventSynchronize(stop));
    float elapsedTime;
    handle_error(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Data Transfer Time: %3.lf ms.\n", elapsedTime);
    handle_error(hipEventDestroy(start));
    handle_error(hipEventDestroy(stop));
    /* ---                  --- */
}