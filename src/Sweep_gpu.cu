#include "hip/hip_runtime.h"
#include "Sweep_gpu.cuh"
#include "Sweep_gpu_Helper.cuh"

/* Helpers for CUDA */
void HANDLE_ERROR(hipError_t error);

void InfectSweep_GPU(double t, int run) {

    int n; // Number of people you could potentially infect in your place group, then number of potential spatial infections doled out by cell on other cells.
    int f, f2, cq /* Cell queue */, bm /* Movement restrictions in place */, ci /* Person index */;
    double s; // Household Force Of Infection (FOI) on fellow household member, then place susceptibility, then random number for spatial infections allocation.
    double s2; // Spatial infectiousness, then distance in spatial infections allocation.
    double s3, s3_scaled; // Household, then place infectiousness.
    double s4, s4_scaled; // Place infectiousness (copy of s3 as some code commented out.
    double s5; // Total spatial infectiousness summed over all infectious people in cell.
    double s6;
    double seasonality, sbeta, hbeta;
    double fp; // False positive.
    unsigned short int ts;

    // If not doing seasonality:
    if (!P.DoSeasonality) {
        // Set seasonality to 1.
        seasonality = 1.0;
    } else {
        // Otherwise pick seasonality from P.Seasonality array using day number in year.
        seasonality = P.Seasonality[((int) t) % DAYS_PER_YEAR];
    }
    // ts = the timestep number of the start of the current day
    ts = (unsigned short int) (P.TimeStepsPerDay * t);
    // fp = false positive
    fp = P.TimeStep / (1 - P.FalsePositiveRate);
    // sbeta seasonality beta
    sbeta = seasonality * fp * P.LocalBeta;
    // hbeta = household beta
    // if doing households, hbeta = seasonality * fp * P.HouseholdTrans, else hbeta = 0
    hbeta = (P.DoHouseholds) ? (seasonality * fp * P.HouseholdTrans) : 0;
    // Establish if movement restrictions are in place on current day - store in bm, 0:false, 1:true
    bm = ((P.DoBlanketMoveRestr) && (t >= P.MoveRestrTimeStart) && (t < P.MoveRestrTimeStart + P.MoveRestrDuration));
    // File for storing error reports
    FILE *stderr_shared = stderr;

#pragma omp parallel for private(n, f, f2, s, s2, s3, s4, s5, s6, cq, ci, s3_scaled, s4_scaled) schedule(static, 1) default(none) \
        shared(t, P, CellLookup, Hosts, AdUnits, Households, Places, SamplingQueue, Cells, Mcells, StateT, hbeta, sbeta, seasonality, ts, fp, bm, stderr_shared)
    for (int tn = 0; tn < P.NumThreads; tn++)
        for (int b = tn; b < P.NCP; b += P.NumThreads) // Loop over (in parallel) all populated cells.
        {
            Cell *c = CellLookup[b]; // Select Cell given by index b.
            s5 = 0; // Spatial infectiousness summed over all infectious people in loop below.

            /* --- Copy Data: Host to Device --- */
//            // Dist global variables:
//            HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(sinx_GPU), sinx, (DEGREES_PER_TURN + 1) * sizeof(double)));
//            HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cosx_GPU), cosx, (DEGREES_PER_TURN + 1) * sizeof(double)));
//            HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(asin2sqx_GPU), asin2sqx, (1001) * sizeof(double)));
//            // Rand global variables:
//            HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(Xcg1_GPU), Xcg1, (MAX_NUM_THREADS * CACHE_LINE_SIZE) * sizeof(int32_t)));
//            HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(Xcg2_GPU), Xcg2, (MAX_NUM_THREADS * CACHE_LINE_SIZE) * sizeof(int32_t)));
            // Cell:
            struct Cell *c_GPU;
            struct Cell *c_Builder = (Cell *) malloc(sizeof(Cell));
            c_Builder->I = c->I;
            HANDLE_ERROR(hipMalloc((void **) &c_Builder->infected, c->I * sizeof(int)));
            HANDLE_ERROR(hipMemcpy(c_Builder->infected, c->infected, c->I * sizeof(int), hipMemcpyHostToDevice));
            HANDLE_ERROR(hipMalloc((void **) &c_GPU, sizeof(struct Cell)));
            HANDLE_ERROR(hipMemcpy(c_GPU, c_Builder, sizeof(struct Cell), hipMemcpyHostToDevice));
            // Hosts:
            struct Person *Hosts_GPU;
            HANDLE_ERROR(hipMalloc((void **) &Hosts_GPU, P.PopSize * sizeof(struct Person)));
            HANDLE_ERROR(hipMemcpy(Hosts_GPU, Hosts, P.PopSize * sizeof(struct Person), hipMemcpyHostToDevice));
            // HostsQuarantine:
            struct PersonQuarantine *HostsQuarantine_GPU;
            HANDLE_ERROR(hipMalloc((void **) &HostsQuarantine_GPU, HostsQuarantine.size() * sizeof(struct PersonQuarantine)));
            HANDLE_ERROR(hipMemcpy(HostsQuarantine_GPU, &HostsQuarantine[0], HostsQuarantine.size() * sizeof(struct PersonQuarantine), hipMemcpyHostToDevice));
            // Households:
            struct Household *Households_GPU;
            HANDLE_ERROR(hipMalloc((void **) &Households_GPU, P.NH * sizeof(struct Household)));
            HANDLE_ERROR(hipMemcpy(Households_GPU, Households, P.NH * sizeof(struct Household), hipMemcpyHostToDevice));
            // Mcells:
            struct Microcell *Mcells_GPU;
            HANDLE_ERROR(hipMalloc((void **) &Mcells_GPU, P.NMC * sizeof(struct Microcell)));
            HANDLE_ERROR(hipMemcpy(Mcells_GPU, Mcells, P.NMC * sizeof(struct Microcell), hipMemcpyHostToDevice));
            // Places:
            struct Place **Struct_Builder = (struct Place **) malloc(P.PlaceTypeNum * sizeof(struct Place *));
            for (int i = 0; i < P.PlaceTypeNum; i++) {
                Struct_Builder[i] = (struct Place *) malloc(P.Nplace[i] * sizeof(struct Place));
                for (int j = 0; j < P.Nplace[i]; j++) {
                    Place place = Places[i][j];
                    Struct_Builder[i][j] = place;
                    HANDLE_ERROR(hipMalloc((void **) &Struct_Builder[i][j].group_start, place.ng * sizeof(int)));
                    HANDLE_ERROR(hipMemcpy(Struct_Builder[i][j].group_start, place.group_start, place.ng * sizeof(int), hipMemcpyHostToDevice));
                    HANDLE_ERROR(hipMalloc((void **) &Struct_Builder[i][j].group_size, place.ng * sizeof(int)));
                    HANDLE_ERROR(hipMemcpy(Struct_Builder[i][j].group_size, place.group_size, place.ng * sizeof(int), hipMemcpyHostToDevice));
                    if (i == P.HotelPlaceType) {
                        HANDLE_ERROR(hipMalloc((void **) &Struct_Builder[i][j].members, 2 * ((int)P.PlaceTypeMeanSize[i]) * sizeof(int)));
                        HANDLE_ERROR(hipMemcpy(Struct_Builder[i][j].members, place.members, 2 * ((int)P.PlaceTypeMeanSize[i]) * sizeof(int), hipMemcpyHostToDevice));
                    } else {
                        HANDLE_ERROR(hipMalloc((void **) &Struct_Builder[i][j].members, place.n * sizeof(int)));
                        HANDLE_ERROR(hipMemcpy(Struct_Builder[i][j].members, place.members, place.n * sizeof(int), hipMemcpyHostToDevice));
                    }
                }
            }
            struct Place **Places_GPU;
            struct Place *Places_Builder[P.PlaceTypeNum];
            HANDLE_ERROR(hipMalloc((void **) &Places_GPU, P.PlaceTypeNum * sizeof(struct Place *)));
            for (int i = 0; i < P.PlaceTypeNum; i++) {
                HANDLE_ERROR(hipMalloc((void **) &Places_Builder[i], P.Nplace[i] * sizeof(struct Place)));
                HANDLE_ERROR(hipMemcpy(Places_Builder[i], Struct_Builder[i], P.Nplace[i] * sizeof(struct Place), hipMemcpyHostToDevice));
            }
            HANDLE_ERROR(hipMemcpy(Places_GPU, Places_Builder, P.PlaceTypeNum * sizeof(struct Place *), hipMemcpyHostToDevice));
            // AdUnits:
            struct AdminUnit *AdUnits_GPU;
            HANDLE_ERROR(hipMalloc((void **) &AdUnits_GPU, MAX_ADUNITS * sizeof(struct AdminUnit)));
            HANDLE_ERROR(hipMemcpy(AdUnits_GPU, AdUnits, MAX_ADUNITS * sizeof(struct AdminUnit), hipMemcpyHostToDevice));
            // SamplingQueue:
            int **SamplingQueue_GPU;
            int *SamplingQueue_Builder[P.NumThreads];
            HANDLE_ERROR(hipMalloc((void **) &SamplingQueue_GPU, P.NumThreads * sizeof(int *)));
            for (int i = 0; i < P.NumThreads; i++) {
                HANDLE_ERROR(hipMalloc((void **) &SamplingQueue_Builder[i], 2 * (MAX_PLACE_SIZE + CACHE_LINE_SIZE) * sizeof(int)));
                HANDLE_ERROR(hipMemcpy(SamplingQueue_Builder[i], SamplingQueue[i], 2 * (MAX_PLACE_SIZE + CACHE_LINE_SIZE) * sizeof(int), hipMemcpyHostToDevice));
            }
            HANDLE_ERROR(hipMemcpy(SamplingQueue_GPU, SamplingQueue_Builder, P.NumThreads * sizeof(int *), hipMemcpyHostToDevice));
            // StateT:
            struct PopVar *StateT_GPU;
            struct PopVar *StateT_Builder = (struct PopVar *) malloc(P.NumThreads * sizeof(struct PopVar));
            memcpy(StateT_Builder, StateT, P.NumThreads * sizeof(struct PopVar));
            for (int i = 0; i < P.NumThreads; i++) {
                for (int j = 0; j < P.NumThreads; j++) {
                    HANDLE_ERROR(hipMalloc((void **) &(StateT_Builder[i].inf_queue[j]),StateT[i].n_queue[j] * sizeof(Infection)));
                    HANDLE_ERROR(hipMemcpy(StateT_Builder[i].inf_queue[j], StateT[i].inf_queue[j],StateT[i].n_queue[j] * sizeof(Infection),hipMemcpyHostToDevice));
                }
                HANDLE_ERROR(hipMalloc((void **) &StateT_Builder[i].cell_inf, StateT[i].cell_inf_length * sizeof(float)));
                HANDLE_ERROR(hipMemcpy(StateT_Builder[i].cell_inf, StateT[i].cell_inf, StateT[i].cell_inf_length * sizeof(float),hipMemcpyHostToDevice));
                for (int j = 0; j < P.NumAdunits; j++) {
                    HANDLE_ERROR(hipMalloc((void **) &(StateT_Builder[i].dct_queue[j]), StateT[i].ndct_queue[j] * sizeof(ContactEvent)));
                    HANDLE_ERROR(hipMemcpy(StateT_Builder[i].dct_queue[j], StateT[i].dct_queue[j], StateT[i].ndct_queue[j] * sizeof(ContactEvent), hipMemcpyHostToDevice));
                }
            }
            HANDLE_ERROR(hipMalloc((void **) &StateT_GPU, P.NumThreads * sizeof(struct PopVar)));
            HANDLE_ERROR(hipMemcpy(StateT_GPU, StateT_Builder, P.NumThreads * sizeof(struct PopVar),hipMemcpyHostToDevice));
            // P:
            struct Param *P_GPU;
            HANDLE_ERROR(hipMalloc((void **) &P_GPU, sizeof(struct Param)));
            HANDLE_ERROR(hipMemcpy(P_GPU, &P, sizeof(struct Param), hipMemcpyHostToDevice));
            // Data:
            struct Data *data;
            HANDLE_ERROR(hipMalloc((void **) &data, sizeof(struct Data)));
            struct Data *h_data = (struct Data *) malloc(sizeof(struct Data));
            h_data->bm = bm;
            h_data->s5 = s5;
            h_data->seasonality = seasonality;
            h_data->sbeta = sbeta;
            h_data->hbeta = hbeta;
            h_data->fp = fp;
            h_data->ts = ts;
            h_data->need_exit = false;
            h_data->exit_num = 0;
            HANDLE_ERROR(hipMemcpy(data, h_data, sizeof(struct Data), hipMemcpyHostToDevice));
            /* ---                           --- */

            /* --- Start Time Record --- */
            hipEvent_t start, stop;
            HANDLE_ERROR(hipEventCreate(&start));
            HANDLE_ERROR(hipEventCreate(&stop));
            HANDLE_ERROR(hipEventRecord(start, 0));
            /* ---                   --- */

            kernel<<<1, 1>>>(t, tn, c_GPU, Hosts_GPU, HostsQuarantine_GPU, Households_GPU, Mcells_GPU, Places_GPU, AdUnits_GPU, SamplingQueue, StateT_GPU, P_GPU, data);

            /* --- Stop Time Record --- */
            HANDLE_ERROR(hipEventRecord(stop, 0));
            HANDLE_ERROR(hipEventSynchronize(stop));
            float elapsedTime;
            HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
            printf("Infect Sweep Time: %3.lf ms.\n", elapsedTime);
            HANDLE_ERROR(hipEventDestroy(start));
            HANDLE_ERROR(hipEventDestroy(stop));
            /* ---                  --- */

            /* --- Copy Data: Device to Host & Free Memory --- */
            // Cell:
            HANDLE_ERROR(hipFree(c_Builder->infected));
            HANDLE_ERROR(hipFree(c_GPU));
            free(c_Builder);
            // Hosts:
            HANDLE_ERROR(hipMemcpy(Hosts, Hosts_GPU, P.PopSize * sizeof(struct Person), hipMemcpyDeviceToHost));
            HANDLE_ERROR(hipFree(Hosts_GPU));
            // HostsQuarantine:
            HANDLE_ERROR(hipFree(HostsQuarantine_GPU));
            // Households:
            HANDLE_ERROR(hipFree(Households_GPU));
            // Mcells:
            HANDLE_ERROR(hipFree(Mcells_GPU));
            // Places:
            for (int i = 0; i < P.PlaceTypeNum; i++) {
                for (int j = 0; j < P.Nplace[i]; n++) {
                    HANDLE_ERROR(hipFree(Struct_Builder[i][j].group_start));
                    HANDLE_ERROR(hipFree(Struct_Builder[i][j].group_size));
                    HANDLE_ERROR(hipFree(Struct_Builder[i][j].members));
                }
                free(Struct_Builder[i]);
            }
            free(Struct_Builder);
            for (int i = 0; i < P.PlaceTypeNum; i++) {
                HANDLE_ERROR(hipFree(Places_Builder[i]));
            }
            HANDLE_ERROR(hipFree(Places_GPU));
            // AdUnits:
            HANDLE_ERROR(hipFree(AdUnits_GPU));
            // SamplingQueue:
            HANDLE_ERROR(hipMemcpy(SamplingQueue_Builder, SamplingQueue_GPU, P.NumThreads * sizeof(int *),hipMemcpyDeviceToHost));
            for (int i = 0; i < P.NumThreads; i++) {
                HANDLE_ERROR(hipMemcpy(SamplingQueue[i], SamplingQueue_Builder[i], 2 * (MAX_PLACE_SIZE + CACHE_LINE_SIZE) * sizeof(int), hipMemcpyDeviceToHost));
                HANDLE_ERROR(hipFree(SamplingQueue_Builder[i]));
            }
            hipFree(SamplingQueue_GPU);
            // StateT:
            HANDLE_ERROR(hipMemcpy(StateT_Builder, StateT_GPU, P.NumThreads * sizeof(struct PopVar),hipMemcpyDeviceToHost));
            HANDLE_ERROR(hipFree(StateT_GPU));
            for (int i = 0; i < P.NumThreads; i++) {
                for (int j = 0; j < MAX_NUM_THREADS; j++) {
                    HANDLE_ERROR(hipMemcpy(StateT[i].inf_queue[j], StateT_Builder[i].inf_queue[j], StateT[i].n_queue[j] * sizeof(Infection), hipMemcpyDeviceToHost));
                    HANDLE_ERROR(hipFree(StateT_Builder[i].inf_queue[j]));
                }
                memcpy(StateT[i].n_queue, StateT_Builder[i].n_queue, MAX_NUM_THREADS * sizeof(int));
                HANDLE_ERROR(hipFree(StateT_Builder[i].cell_inf));
                for (int j = 0; j < P.NumAdunits; j++) {
                    HANDLE_ERROR(hipMemcpy(StateT[i].dct_queue[j], StateT_Builder[i].dct_queue[j], StateT[i].ndct_queue[j] * sizeof(ContactEvent), hipMemcpyDeviceToHost));
                    HANDLE_ERROR(hipFree(StateT_Builder[i].dct_queue[j]));
                }
            }
            free(StateT_Builder);
            // P:
            HANDLE_ERROR(hipMemcpy(&P, P_GPU, sizeof(struct Param), hipMemcpyDeviceToHost));
            HANDLE_ERROR(hipFree(P_GPU));
            // Data:
            HANDLE_ERROR(hipMemcpy(h_data, data, sizeof(struct Data), hipMemcpyDeviceToHost));
            HANDLE_ERROR(hipFree(data));
            s5 = h_data->s5;
            if (h_data->need_exit) {
                exit(h_data->exit_num);
            }
            free(h_data);
            /* ---                           --- */

            //// Now allocate spatial infections using Force Of Infection (s5) calculated above
            if (s5 > 0) //// if spatial infectiousness positive
            {

                // decide how many potential cell to cell infections this cell could cause
                n = (int) ignpoi_mt(s5 * sbeta * ((double) c->tot_prob),
                                    tn); //// number people this cell's population might infect elsewhere. poisson random number based on spatial infectiousness s5, sbeta (seasonality) and this cell's "probability" (guessing this is a function of its population and geographical size).
                // i2 = number of infectious people in cell c
                int i2 = c->I;

                if (n >
                    0) //// this block normalises cumulative infectiousness cell_inf by person. s5 is the total cumulative spatial infectiousness. Reason is so that infector can be chosen using ranf_mt, which returns random number between 0 and 1.
                {
                    //// normalise by cumulative spatial infectiousness.
                    for (int j = 0; j < i2 - 1; j++) StateT[tn].cell_inf[j] /= ((float) s5);
                    //// does same as the above loop just a slightly faster calculation. i.e. StateT[tn].cell_inf[i2 - 1] / s5 would equal 1 or -1 anyway.
                    StateT[tn].cell_inf[i2 - 1] = (StateT[tn].cell_inf[i2 - 1] < 0) ? -1.0f : 1.0f;
                }

                //// loop over infections to dole out. roughly speaking, this determines which infectious person in cell c infects which person elsewhere.
                for (int k = 0; k < n; k++) {
                    //// decide on infector ci/si from cell c.
                    int j; // j = index of infector
                    // if only one infectious person in cell
                    if (i2 == 1) {
                        j = 0; // infector index is first in cell (person 0)
                    }
                        // if more than one infectious person in cell pick an infectious person (given by index j)
                        //// roughly speaking, this determines which infectious person in cell c infects which person elsewhere
                    else {
                        int m;
                        s = ranf_mt(tn);    ///// choose random number between 0 and 1
                        j = m = i2 /
                                2;        ///// assign j and m to be halfway between zero and number of infected people i2 = c->I.
                        f = 1;
                        do {
                            if (m > 1)
                                m /= 2; //// amount m to change j by reduced by half. Looks like a binary search. Basically saying, keep amending potential infector j until either j less than zero or more than number of infected people until you find j s.t. spatial infectiousness "matches" s.
                            if ((j > 0) && (fabs(StateT[tn].cell_inf[j - 1]) >= s)) {
                                j -= m;
                                if (j == 0) f = 0;
                            } else if ((j < i2 - 1) && (fabs(StateT[tn].cell_inf[j]) < s)) {
                                j += m;
                                if (j == i2 - 1) f = 0;
                            } else f = 0;
                        } while (f);
                    }
                    f = (StateT[tn].cell_inf[j] <
                         0); //// flag for whether infector j had their place(s) closed. <0 (true) = place closed / >=0 (false) = place not closed. Set in if (sbeta > 0) part of loop over infectious people.
                    // ci is the index of the jth infectious person in the cell
                    ci = c->infected[j];
                    // si is the jth selected person in the cell
                    Person *si = Hosts + ci;

                    //calculate flag (fct) for digital contact tracing here at the beginning for each individual infector
                    int fct = ((P.DoDigitalContactTracing) &&
                               (t >= AdUnits[Mcells[si->mcell].adunit].DigitalContactTracingTimeStart)
                               && (t < AdUnits[Mcells[si->mcell].adunit].DigitalContactTracingTimeStart +
                                       P.DigitalContactTracingPolicyDuration) && (Hosts[ci].digitalContactTracingUser ==
                                                                                  1)); // && (ts <= (Hosts[ci].detected_time + P.usCaseIsolationDelay)));


                    //// decide on infectee

                    // do the following while f2=0
                    do {
                        //// chooses which cell person will infect
                        // pick random s between 0 and 1
                        s = ranf_mt(tn);
                        // generate l using InvCDF of selected cell and random integer between 0 and 1024
                        int l = c->InvCDF[(int) floor(s * 1024)];
                        // loop over c->cum_trans array until find a value >= random number s
                        while (c->cum_trans[l] < s) l++;
                        // selecte the cell corresponding to l
                        Cell *ct = CellLookup[l];

                        ///// pick random person m within susceptibles of cell ct (S0 initial number susceptibles within cell).
                        int m = (int) (ranf_mt(tn) * ((double) ct->S0));
                        int i3 = ct->susceptible[m];

                        s2 = dist2(Hosts + i3, Hosts +
                                               ci); /// calculate distance squared between this susceptible person and person ci/si identified earlier
                        s = P.KernelLookup.num(s2) / c->max_trans[l]; //// acceptance probability

                        // initialise f2=0 (f2=1 is the while condition for this loop)
                        f2 = 0;
                        // if random number greater than acceptance probablility or infectee is dead
                        if ((ranf_mt(tn) >= s) || (abs(Hosts[i3].inf) ==
                                                   InfStat_Dead)) //// if rejected, or infectee i3/m already dead, ensure do-while evaluated again (i.e. choose a new infectee).
                        {
                            // set f2=1 so loop continues
                            f2 = 1;
                        } else {
                            //// if potential infectee not travelling, and either is not part of cell c or doesn't share a household with infector.
                            if ((!Hosts[i3].Travelling) && ((c != ct) || (Hosts[i3].hh != si->hh))) {
                                // pick microcell of infector (mi)
                                Microcell *mi = Mcells + si->mcell;
                                // pick microcell of infectee (mt)
                                Microcell *mt = Mcells + Hosts[i3].mcell;
                                s = CalcSpatialSusc(i3, ts, ci, tn);
                                // Care home residents may have fewer contacts
                                if ((Hosts[i3].care_home_resident) || (Hosts[ci].care_home_resident))
                                    s *= P.CareHomeResidentSpatialScaling;
                                //so this person is a contact - but might not be infected. if we are doing digital contact tracing, we want to add the person to the contacts list, if both are users
                                if (fct) {
                                    //if infectee is also a user, add them as a contact
                                    if (Hosts[i3].digitalContactTracingUser && (ci != i3)) {
                                        if ((Hosts[ci].ncontacts < P.MaxDigitalContactsToTrace) &&
                                            (ranf_mt(tn) < s * P.ProportionDigitalContactsIsolate)) {
                                            Hosts[ci].ncontacts++; //add to number of contacts made
                                            int ad = Mcells[Hosts[i3].mcell].adunit;
                                            if ((StateT[tn].ndct_queue[ad] < AdUnits[ad].n)) {
                                                //find adunit for contact and add both contact and infectious host to lists - storing both so I can set times later.
                                                StateT[tn].dct_queue[ad][StateT[tn].ndct_queue[ad]++] = {i3, ci, ts};
                                            } else {
                                                fprintf(stderr_shared,
                                                        "No more space in queue! Thread: %i, AdUnit: %i\n", tn, ad);
                                            }
                                        }
                                    }
                                    //scale down susceptibility so we don't over accept
                                    s /= P.ScalingFactorSpatialDigitalContacts;
                                }
                                if (m < ct->S)  // only bother trying to infect susceptible people
                                {
                                    s *= CalcPersonSusc(i3, ts, ci, tn);
                                    if (bm) {
                                        if ((dist2_raw(Households[si->hh].loc.x, Households[si->hh].loc.y,
                                                       Households[Hosts[i3].hh].loc.x, Households[Hosts[i3].hh].loc.y) >
                                             P.MoveRestrRadius2))
                                            s *= P.MoveRestrEffect;
                                    } else if ((mt->moverest != mi->moverest) &&
                                               ((mt->moverest == 2) || (mi->moverest == 2)))
                                        s *= P.MoveRestrEffect;
                                    if ((!f) && (HOST_ABSENT(
                                            i3))) //// if infector did not have place closed, loop over place types of infectee i3 to see if their places had closed. If they had, amend their susceptibility.
                                    {
                                        for (m = f2 = 0; (m < P.PlaceTypeNum) && (!f2); m++)
                                            if (Hosts[i3].PlaceLinks[m] >= 0) {
                                                f2 = PLACE_CLOSED(m, Hosts[i3].PlaceLinks[m]);
                                            }
                                        if (f2) { s *= P.PlaceCloseSpatialRelContact; }/* NumPCD++;} */
                                        f2 = 0;
                                    }
                                    if ((s == 1) || (ranf_mt(tn) < s)) //// accept/reject
                                    {
                                        cq = ((int) (ct - Cells)) % P.NumThreads;
                                        if ((Hosts[i3].inf == InfStat_Susceptible) &&
                                            (StateT[tn].n_queue[cq] < P.InfQueuePeakLength)) //Hosts[i3].infector==-1
                                        {
                                            if ((P.FalsePositiveRate > 0) && (ranf_mt(tn) < P.FalsePositiveRate))
                                                StateT[tn].inf_queue[cq][StateT[tn].n_queue[cq]++] = {-1, i3, -1};
                                            else {
                                                short int infect_type = 2 + 2 * NUM_PLACE_TYPES + INFECT_TYPE_MASK *
                                                                                                  (1 + si->infect_type /
                                                                                                       INFECT_TYPE_MASK);
                                                StateT[tn].inf_queue[cq][StateT[tn].n_queue[cq]++] = {ci, i3,
                                                                                                      infect_type};
                                            }
                                        }
                                    }
                                }// m < susceptible people in target cell
                            }// //// if potential infectee not travelling, and either is not part of cell c or doesn't share a household with infector
                        }// infectee isn't dead
                    } while (f2);
                }// loop over infections doled out by cell
            }// s5 > 0
        }


#pragma omp parallel for schedule(static, 1) default(none) \
        shared(t, run, P, StateT, Hosts, ts)
    for (int j = 0; j < P.NumThreads; j++) {
        for (int k = 0; k < P.NumThreads; k++) {
            for (int i = 0; i < StateT[k].n_queue[j]; i++) {
                int infector = StateT[k].inf_queue[j][i].infector;
                int infectee = StateT[k].inf_queue[j][i].infectee;
                short int infect_type = StateT[k].inf_queue[j][i].infect_type;
                Hosts[infectee].infector = infector;
                Hosts[infectee].infect_type = infect_type;
                if (infect_type == -1) //// i.e. if host doesn't have an infector
                    DoFalseCase(infectee, t, ts, j);
                else
                    DoInfect(infectee, t, j, run);
            }
            StateT[k].n_queue[j] = 0;
        }
    }
}


// Error Handling.
void HANDLE_ERROR(hipError_t error) {
    if (error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
}
